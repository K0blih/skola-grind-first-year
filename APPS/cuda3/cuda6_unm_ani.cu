#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage without unified memory.
//
// Simple animation with image rotation.
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "cuda_img.h"
#include "animation.h"

// Demo kernel to create chess board
__global__ void kernel_createGradient(CudaImg colorCudaImg)
{
    // X,Y coordinates and check image dimensions
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (y >= colorCudaImg.m_size.y) return;
    if (x >= colorCudaImg.m_size.x) return;

    int dy = x * colorCudaImg.m_size.y / colorCudaImg.m_size.x + y - colorCudaImg.m_size.y;
    unsigned char color = 255 * abs(dy) / colorCudaImg.m_size.y;

    uchar3 bgr = (dy < 0) ? (uchar3){color, 255 - color, 0} : (uchar3){0, 255 - color, color};

    // Store point into image
    colorCudaImg.m_p_uchar3[y * colorCudaImg.m_size.x + x] = bgr;
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_insertImage(CudaImg bigCudaImg, CudaImg smallCudaPic, int2 position)
{
    // X,Y coordinates and check image dimensions
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (y >= smallCudaPic.m_size.y) return;
    if (x >= smallCudaPic.m_size.x) return;
    int by = y + position.y;
    int bx = x + position.x;
    if (by >= bigCudaImg.m_size.y || by < 0) return;
    if (bx >= bigCudaImg.m_size.x || bx < 0) return;

    // Get point from small image
    uchar4 fgBgra = smallCudaPic.m_p_uchar4[y * smallCudaPic.m_size.x + x];
    uchar3 bgBgr = bigCudaImg.m_p_uchar3[by * bigCudaImg.m_size.x + bx];
    uchar3 bgr = {0, 0, 0};

    // compose point from small and big image according alpha channel
    bgr.x = fgBgra.x * fgBgra.w / 255 + bgBgr.x * (255 - fgBgra.w) / 255;
    bgr.y = fgBgra.y * fgBgra.w / 255 + bgBgr.y * (255 - fgBgra.w) / 255;
    bgr.z = fgBgra.z * fgBgra.w / 255 + bgBgr.z * (255 - fgBgra.w) / 255;

    // Store point into image
    bigCudaImg.m_p_uchar3[by * bigCudaImg.m_size.x + bx] = bgr;
}

// New kernel for image rotation
__global__ void kernel_rotateImage(CudaImg origCudaImg, CudaImg rotateCudaImg, float sinAngle, float cosAngle)
{
    // X,Y coordinates and check image dimensions
    int rotateY = blockDim.y * blockIdx.y + threadIdx.y;
    int rotateX = blockDim.x * blockIdx.x + threadIdx.x;
    if (rotateY >= rotateCudaImg.m_size.y) return;
    if (rotateX >= rotateCudaImg.m_size.x) return;

    // Recalculation from image coordinates to centerpoint coordinates
    int crotateX = rotateX - rotateCudaImg.m_size.x / 2;
    int crotateY = rotateY - rotateCudaImg.m_size.y / 2;

    // Position in original image
    float corigX = cosAngle * crotateX - sinAngle * crotateY;
    float corigY = sinAngle * crotateX + cosAngle * crotateY;
    
    // Recalculation from centerpoint coordinates to image coordinates
    int origX = corigX + origCudaImg.m_size.x / 2;
    int origY = corigY + origCudaImg.m_size.y / 2;

    // Check if within original image boundaries
    if (origX < 0 || origX >= origCudaImg.m_size.x) return;
    if (origY < 0 || origY >= origCudaImg.m_size.y) return;

    // Copy pixel from original image to rotated image
    rotateCudaImg.m_p_uchar4[rotateY * rotateCudaImg.m_size.x + rotateX] = origCudaImg.m_p_uchar4[origY * origCudaImg.m_size.x + origX];
}

void cu_insertImage(CudaImg bigCudaImg, CudaImg smallCudaPic, int2 position)
{
    hipError_t cerr;

    // Grid creation, size of grid must be equal or greater than images
    int blockSize = 32;
    dim3 blocks((smallCudaPic.m_size.x + blockSize - 1) / blockSize,
                (smallCudaPic.m_size.y + blockSize - 1) / blockSize);
    dim3 threads(blockSize, blockSize);
    kernel_insertImage<<<blocks, threads>>>(bigCudaImg, smallCudaPic, position);

    if ((cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    hipDeviceSynchronize();
}

// New function for rotating images
void cu_rotateImage(CudaImg origCudaImg, CudaImg rotateCudaImg, float angle)
{
    hipError_t cerr;

	float angleRad = angle * 3.14159265f / 180.0f;

    // Pre-calculate sin and cos values for the kernel
    float sinAngle = sinf(angleRad);
    float cosAngle = cosf(angleRad);

    // Grid creation, size of grid must be equal or greater than rotated image
    int blockSize = 32;
    dim3 blocks((rotateCudaImg.m_size.x + blockSize - 1) / blockSize,
                (rotateCudaImg.m_size.y + blockSize - 1) / blockSize);
    dim3 threads(blockSize, blockSize);
    
    // Call kernel with pre-calculated sin/cos values
    kernel_rotateImage<<<blocks, threads>>>(origCudaImg, rotateCudaImg, sinAngle, cosAngle);

    if ((cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    hipDeviceSynchronize();
}

__global__ void kernel_scale_4ch(CudaImg t_img_in, CudaImg t_img_out)
{
    // X,Y coordinates and check output image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_img_out.m_size.y) return;
    if (l_x >= t_img_out.m_size.x) return;

    // Calculate scaling factors
    float l_scale_x = (float)t_img_in.m_size.x / t_img_out.m_size.x;
    float l_scale_y = (float)t_img_in.m_size.y / t_img_out.m_size.y;
    
    // Calculate source position in original image
    float l_orig_x = l_x * l_scale_x;
    float l_orig_y = l_y * l_scale_y;
    
    // Get the four surrounding pixels
    int l_x0 = (int)l_orig_x;
    int l_y0 = (int)l_orig_y;
    int l_x1 = min(l_x0 + 1, (int)t_img_in.m_size.x - 1);
    int l_y1 = min(l_y0 + 1, (int)t_img_in.m_size.y - 1);
    
    // Calculate interpolation weights
    float l_dx = l_orig_x - l_x0;
    float l_dy = l_orig_y - l_y0;
    
    // Get the four surrounding colors using at4
    uchar4 l_c00 = t_img_in.at4(l_y0, l_x0);
    uchar4 l_c10 = t_img_in.at4(l_y0, l_x1);
    uchar4 l_c01 = t_img_in.at4(l_y1, l_x0);
    uchar4 l_c11 = t_img_in.at4(l_y1, l_x1);
    
    // Interpolate to get the output color
    uchar4 l_result;
    
    // Interpolate for each channel (RGB + alpha)
    // R channel
    float l_s0_r = l_c00.x * (1 - l_dx) + l_c10.x * l_dx;
    float l_s1_r = l_c01.x * (1 - l_dx) + l_c11.x * l_dx;
    l_result.x = (unsigned char)(l_s0_r * (1 - l_dy) + l_s1_r * l_dy);
    
    // G channel
    float l_s0_g = l_c00.y * (1 - l_dx) + l_c10.y * l_dx;
    float l_s1_g = l_c01.y * (1 - l_dx) + l_c11.y * l_dx;
    l_result.y = (unsigned char)(l_s0_g * (1 - l_dy) + l_s1_g * l_dy);
    
    // B channel
    float l_s0_b = l_c00.z * (1 - l_dx) + l_c10.z * l_dx;
    float l_s1_b = l_c01.z * (1 - l_dx) + l_c11.z * l_dx;
    l_result.z = (unsigned char)(l_s0_b * (1 - l_dy) + l_s1_b * l_dy);
    
    // Alpha channel
    float l_s0_a = l_c00.w * (1 - l_dx) + l_c10.w * l_dx;
    float l_s1_a = l_c01.w * (1 - l_dx) + l_c11.w * l_dx;
    l_result.w = (unsigned char)(l_s0_a * (1 - l_dy) + l_s1_a * l_dy);
    
    // Store result in output image using at4
    t_img_out.at4(l_y, l_x) = l_result;
}

// Kernel to scale a 3-channel image using bilinear interpolation
__global__ void kernel_scale(CudaImg t_img_in, CudaImg t_img_out)
{
    // X,Y coordinates and check output image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_img_out.m_size.y) return;
    if (l_x >= t_img_out.m_size.x) return;

    // Calculate scaling factors
    float l_scale_x = (float)t_img_in.m_size.x / t_img_out.m_size.x;
    float l_scale_y = (float)t_img_in.m_size.y / t_img_out.m_size.y;
    
    // Calculate source position in original image
    float l_orig_x = l_x * l_scale_x;
    float l_orig_y = l_y * l_scale_y;
    
    // Get the four surrounding pixels
    int l_x0 = (int)l_orig_x;
    int l_y0 = (int)l_orig_y;
    int l_x1 = min(l_x0 + 1, (int)t_img_in.m_size.x - 1);
    int l_y1 = min(l_y0 + 1, (int)t_img_in.m_size.y - 1);
    
    // Calculate interpolation weights
    float l_dx = l_orig_x - l_x0;
    float l_dy = l_orig_y - l_y0;
    
    // Get the four surrounding colors using at3
    uchar3 l_c00 = t_img_in.at3(l_y0, l_x0);
    uchar3 l_c10 = t_img_in.at3(l_y0, l_x1);
    uchar3 l_c01 = t_img_in.at3(l_y1, l_x0);
    uchar3 l_c11 = t_img_in.at3(l_y1, l_x1);
    
    // Interpolate to get the output color
    uchar3 l_result;
    
    // Interpolate for each channel (RGB)
    // R channel
    float l_s0_r = l_c00.x * (1 - l_dx) + l_c10.x * l_dx;
    float l_s1_r = l_c01.x * (1 - l_dx) + l_c11.x * l_dx;
    l_result.x = (unsigned char)(l_s0_r * (1 - l_dy) + l_s1_r * l_dy);
    
    // G channel
    float l_s0_g = l_c00.y * (1 - l_dx) + l_c10.y * l_dx;
    float l_s1_g = l_c01.y * (1 - l_dx) + l_c11.y * l_dx;
    l_result.y = (unsigned char)(l_s0_g * (1 - l_dy) + l_s1_g * l_dy);
    
    // B channel
    float l_s0_b = l_c00.z * (1 - l_dx) + l_c10.z * l_dx;
    float l_s1_b = l_c01.z * (1 - l_dx) + l_c11.z * l_dx;
    l_result.z = (unsigned char)(l_s0_b * (1 - l_dy) + l_s1_b * l_dy);
    
    // Store result in output image using at3
    t_img_out.at3(l_y, l_x) = l_result;
}

// Function to handle the image scaling with channel detection
void cu_scale(CudaImg &t_cu_orig, CudaImg &t_cu_scaled)
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than output image
    int l_block_size = 32;
    dim3 l_blocks((t_cu_scaled.m_size.x + l_block_size - 1) / l_block_size,
                 (t_cu_scaled.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    
    // Check if we're dealing with a 3-channel or 4-channel image
    bool is_4ch = (t_cu_orig.m_p_uchar4 != nullptr);
    
    if (is_4ch) {
        kernel_scale_4ch<<<l_blocks, l_threads>>>(t_cu_orig, t_cu_scaled);
    } else {
        kernel_scale<<<l_blocks, l_threads>>>(t_cu_orig, t_cu_scaled);
    }

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}
// -----------------------------------------------------------------------------------------------

void Animation::start(CudaImg bgCudaImg, CudaImg insCudaImg)
{
    if (m_initialized) return;
    hipError_t cerr;

    m_bg_cuda_img = bgCudaImg;
    m_res_cuda_img = bgCudaImg;
    m_ins_cuda_img = insCudaImg;
    
    // Allocate memory for rotated ball image
    m_rot_cuda_img.m_size = insCudaImg.m_size;
    cerr = hipMalloc(&m_rot_cuda_img.m_p_void, m_rot_cuda_img.m_size.x * m_rot_cuda_img.m_size.y * sizeof(uchar4));
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    // Memory allocation in GPU device
    // Memory for background
    cerr = hipMalloc(&m_bg_cuda_img.m_p_void, m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3));
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    // Memory for result
    cerr = hipMalloc(&m_res_cuda_img.m_p_void, m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof(uchar3));
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    // Creation of background gradient
    int blockSize = 32;
    dim3 blocks((m_bg_cuda_img.m_size.x + blockSize - 1) / blockSize,
                (m_bg_cuda_img.m_size.y + blockSize - 1) / blockSize);
    dim3 threads(blockSize, blockSize);
    kernel_createGradient<<<blocks, threads>>>(m_bg_cuda_img);

    m_initialized = 1;
}

void Animation::next(CudaImg resCudaImg, int2 position, float angleDeg)
{
    if (!m_initialized) return;

    hipError_t cerr;

    // Copy data internally GPU from background into result
    cerr = hipMemcpy(m_res_cuda_img.m_p_void, m_bg_cuda_img.m_p_void, 
                     m_bg_cuda_img.m_size.x * m_bg_cuda_img.m_size.y * sizeof(uchar3), 
                     hipMemcpyDeviceToDevice);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    // Rotate the ball based on its position
    // static float rotationAngle = 0.0f;
    // rotationAngle = position.x * 0.01f; // Adjust rotation speed based on X position
    
    // Rotate the image
    cu_rotateImage(m_ins_cuda_img, m_rot_cuda_img, angleDeg);
    
    // Insert the rotated image
    cu_insertImage(m_res_cuda_img, m_rot_cuda_img, position);

    // Copy data to GPU device
    cerr = hipMemcpy(resCudaImg.m_p_void, m_res_cuda_img.m_p_void, 
                     m_res_cuda_img.m_size.x * m_res_cuda_img.m_size.y * sizeof(uchar3), 
                     hipMemcpyDeviceToHost);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));
}

void Animation::stop()
{
    if (!m_initialized) return;

    hipFree(m_bg_cuda_img.m_p_void);
    hipFree(m_res_cuda_img.m_p_void);
    hipFree(m_ins_cuda_img.m_p_void);
    hipFree(m_rot_cuda_img.m_p_void); // Free memory for rotated image

    m_initialized = 0;
}