#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage with unified memory.
//
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

// Demo kernel to transform RGB color schema to BW schema
__global__ void kernel_grayscale( CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    // Get point from color picture
    uchar3 l_bgr = t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ];

    // Store BW point to new image
    t_bw_cuda_img.m_p_uchar1[ l_y * t_bw_cuda_img.m_size.x + l_x ].x = l_bgr.x * 0.11 + l_bgr.y * 0.59 + l_bgr.z * 0.30;
}

void cu_run_grayscale( CudaImg t_color_cuda_img, CudaImg t_bw_cuda_img )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 16;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size, ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_grayscale<<< l_blocks, l_threads >>>( t_color_cuda_img, t_bw_cuda_img );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

__global__ void kernel_insert_image(CudaImg t_big_img, CudaImg t_small_img, int2 t_pos)
{
    // Calculate global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if within small image bounds
    if (x < t_small_img.m_size.x && y < t_small_img.m_size.y) 
    {
        // Calculate target position in big image
        int targetX = x + t_pos.x;
        int targetY = y + t_pos.y;
        
        // Check if target position is within big image bounds
        if (targetX < t_big_img.m_size.x && targetY < t_big_img.m_size.y) 
        {
            // Copy pixel from small image to big image
            // We'll handle both RGB and grayscale cases
            if (t_big_img.m_p_uchar3 && t_small_img.m_p_uchar3) {
                // RGB to RGB
                t_big_img.at3(targetY, targetX) = t_small_img.at3(y, x);
            }
            else if (t_big_img.m_p_uchar1 && t_small_img.m_p_uchar1) {
                // Grayscale to Grayscale
                t_big_img.at1(targetY, targetX) = t_small_img.at1(y, x);
            }
            else if (t_big_img.m_p_uchar3 && t_small_img.m_p_uchar1) {
                // Grayscale to RGB
                uchar1 pixel = t_small_img.at1(y, x);
                uchar3 color;
                color.x = color.y = color.z = pixel.x;
                t_big_img.at3(targetY, targetX) = color;
            }
        }
    }
}

// Function to be called from main program
void cu_insert_image(CudaImg t_cuda_big_img, CudaImg t_cuda_small_img, int2 t_pos)
{
    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (t_cuda_small_img.m_size.x + blockSize.x - 1) / blockSize.x,
        (t_cuda_small_img.m_size.y + blockSize.y - 1) / blockSize.y
    );
    
    // Launch the kernel
    kernel_insert_image<<<gridSize, blockSize>>>(t_cuda_big_img, t_cuda_small_img, t_pos);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
}

__global__ void kernel_insert_image_with_mask(CudaImg t_big_img, CudaImg t_small_img, int2 t_position, uchar3 t_mask)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (l_y >= t_small_img.m_size.y || l_x >= t_small_img.m_size.x)
        return;
    
    // Calculate position in the big image
    int big_y = t_position.y + l_y;
    int big_x = t_position.x + l_x;
    
    // Check if the position is within the big image boundaries
    if (big_y >= t_big_img.m_size.y || big_x >= t_big_img.m_size.x)
        return;
    
    // Get the pixel values
    uchar3 small_pixel = t_small_img.at3(l_y, l_x);
    uchar3 big_pixel = t_big_img.at3(big_y, big_x);
    
    // Apply mask (either 0/1 multiplication or 0/255 AND operation)
    uchar3 result;
    
    // Check if mask is binary (0/1) or byte (0/255)
    if (t_mask.x <= 1 && t_mask.y <= 1 && t_mask.z <= 1) {
        // Using multiplication for 0/1 mask
        result.x = big_pixel.x * (1 - t_mask.x) + small_pixel.x * t_mask.x;
        result.y = big_pixel.y * (1 - t_mask.y) + small_pixel.y * t_mask.y;
        result.z = big_pixel.z * (1 - t_mask.z) + small_pixel.z * t_mask.z;
    } else {
        // Using AND for 0/255 mask
        result.x = (big_pixel.x & ~t_mask.x) | (small_pixel.x & t_mask.x);
        result.y = (big_pixel.y & ~t_mask.y) | (small_pixel.y & t_mask.y);
        result.z = (big_pixel.z & ~t_mask.z) | (small_pixel.z & t_mask.z);
    }
    
    // Save the result
    t_big_img.at3(big_y, big_x) = result;
}

// Function to insert image with color mask - runs on CPU
void cu_insert_image_with_mask(CudaImg t_cuda_big_img, CudaImg t_cuda_small_img, int2 t_pos, uchar3 t_mask)
{
    // Grid creation with calculated size
    hipError_t l_cerr;
    
    // Grid size
    dim3 l_blocks((t_cuda_small_img.m_size.x + 15) / 16, (t_cuda_small_img.m_size.y + 15) / 16);
    dim3 l_threads(16, 16);
    
    // Call the kernel
    kernel_insert_image_with_mask<<<l_blocks, l_threads>>>(t_cuda_big_img, t_cuda_small_img, t_pos, t_mask);
    
    // Wait for all threads to complete
    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));
    
    hipDeviceSynchronize();
}

// Implementation of image swap kernel (swaps regions between two images)
__global__ void kernel_swap_image_quadrants(CudaImg t_dest_img, CudaImg t_src_img1, CudaImg t_src_img2, int2 t_quadrant1, int2 t_quadrant2)
{
    // X,Y coordinates
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Calculate the quadrant size (half of each source image)
    int quad_width = t_src_img1.m_size.x / 2;
    int quad_height = t_src_img1.m_size.y / 2;
    
    if (l_y >= quad_height || l_x >= quad_width)
        return;
    
    // Calculate source positions
    int src1_y = t_quadrant1.y * quad_height + l_y;
    int src1_x = t_quadrant1.x * quad_width + l_x;
    
    int src2_y = t_quadrant2.y * quad_height + l_y;
    int src2_x = t_quadrant2.x * quad_width + l_x;
    
    // Calculate destination positions (swap quarters)
    int dest1_y = t_quadrant1.y * quad_height + l_y;
    int dest1_x = t_quadrant1.x * quad_width + l_x;
    
    int dest2_y = t_quadrant2.y * quad_height + l_y;
    int dest2_x = t_quadrant2.x * quad_width + l_x;
    
    // Perform the swap (copy source 1 to destination 2, source 2 to destination 1)
    t_dest_img.at3(dest2_y, dest2_x) = t_src_img1.at3(src1_y, src1_x);
    t_dest_img.at3(dest1_y, dest1_x) = t_src_img2.at3(src2_y, src2_x);
}

// Function to swap image quadrants - runs on CPU
void cu_swap_image_quadrants(CudaImg t_dest_img, CudaImg t_src_img1, CudaImg t_src_img2, int2 t_quadrant1, int2 t_quadrant2)
{
    // Grid creation with calculated size
    hipError_t l_cerr;
    
    // Calculate quadrant size
    int quad_width = t_src_img1.m_size.x / 2;
    int quad_height = t_src_img1.m_size.y / 2;
    
    // Grid size (for one quadrant)
    dim3 l_blocks((quad_width + 15) / 16, (quad_height + 15) / 16);
    dim3 l_threads(16, 16);
    
    // Call the kernel
    kernel_swap_image_quadrants<<<l_blocks, l_threads>>>(t_dest_img, t_src_img1, t_src_img2, t_quadrant1, t_quadrant2);
    
    // Wait for all threads to complete
    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));
    
    hipDeviceSynchronize();
}

__global__ void kernel_resize_nn_uchar3(uchar3* input, uchar3* output, int inW, int inH, int outW, int outH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < outW && y < outH) {
        float srcX = (float)x * inW / outW;
        float srcY = (float)y * inH / outH;
        int ix = min(int(srcX), inW - 1);
        int iy = min(int(srcY), inH - 1);
        output[y * outW + x] = input[iy * inW + ix];
    }
}

__global__ void kernel_resize_nn_uchar1(uchar1* input, uchar1* output, int inW, int inH, int outW, int outH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < outW && y < outH) {
        float srcX = (float)x * inW / outW;
        float srcY = (float)y * inH / outH;
        int ix = min(int(srcX), inW - 1);
        int iy = min(int(srcY), inH - 1);
        output[y * outW + x].x = input[iy * inW + ix].x;
    }
}

__global__ void kernel_resize_bilinear_uchar3(uchar3* input, uchar3* output, int inW, int inH, int outW, int outH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outW && y < outH) {
        float gx = (float)x * inW / outW;
        float gy = (float)y * inH / outH;
        int ix = floor(gx);
        int iy = floor(gy);
        float dx = gx - ix;
        float dy = gy - iy;
        ix = min(ix, inW - 2);
        iy = min(iy, inH - 2);
        uchar3 c00 = input[iy * inW + ix];
        uchar3 c10 = input[iy * inW + ix + 1];
        uchar3 c01 = input[(iy + 1) * inW + ix];
        uchar3 c11 = input[(iy + 1) * inW + ix + 1];
        float3 top = make_float3(c00.x + dx * (c10.x - c00.x), c00.y + dx * (c10.y - c00.y), c00.z + dx * (c10.z - c00.z));
        float3 bottom = make_float3(c01.x + dx * (c11.x - c01.x), c01.y + dx * (c11.y - c01.y), c01.z + dx * (c11.z - c01.z));
        float3 value = make_float3(top.x + dy * (bottom.x - top.x), top.y + dy * (bottom.y - top.y), top.z + dy * (bottom.z - top.z));
        output[y * outW + x] = make_uchar3((unsigned char)value.x, (unsigned char)value.y, (unsigned char)value.z);
    }
    }

__global__ void kernel_resize_bilinear_uchar1(uchar1* input, uchar1* output, int inW, int inH, int outW, int outH) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outW && y < outH) {
        float gx = (float)x * inW / outW;
        float gy = (float)y * inH / outH;
        int ix = floor(gx);
        int iy = floor(gy);
        float dx = gx - ix;
        float dy = gy - iy;
        ix = min(ix, inW - 2);
        iy = min(iy, inH - 2);
        unsigned char v00 = input[iy * inW + ix].x;
        unsigned char v10 = input[iy * inW + ix + 1].x;
        unsigned char v01 = input[(iy + 1) * inW + ix].x;
        unsigned char v11 = input[(iy + 1) * inW + ix + 1].x;
        float top = v00 + dx * (v10 - v00);
        float bottom = v01 + dx * (v11 - v01);
        unsigned char value = (unsigned char)(top + dy * (bottom - top));
        output[y * outW + x].x = value;
    }
}

void cu_resize(const CudaImg& in_img, CudaImg& out_img, bool bilinear) {
    int inW = in_img.m_size.x;
    int inH = in_img.m_size.y;
    int outW = out_img.m_size.x;
    int outH = out_img.m_size.y;
    dim3 blockDim(16, 16);
    dim3 gridDim((outW + blockDim.x - 1) / blockDim.x, (outH + blockDim.y - 1) / blockDim.y);

    if (bilinear) {
        if (in_img.m_p_uchar3 && out_img.m_p_uchar3) {
            kernel_resize_bilinear_uchar3<<<gridDim, blockDim>>>(in_img.m_p_uchar3, out_img.m_p_uchar3, inW, inH, outW, outH);
        } 
        else if (in_img.m_p_uchar1 && out_img.m_p_uchar1) {
            kernel_resize_bilinear_uchar1<<<gridDim, blockDim>>>(in_img.m_p_uchar1, out_img.m_p_uchar1, inW, inH, outW, outH);
        }
    } 
    else {
        if (in_img.m_p_uchar3 && out_img.m_p_uchar3) {
            kernel_resize_nn_uchar3<<<gridDim, blockDim>>>(in_img.m_p_uchar3, out_img.m_p_uchar3,inW, inH, outW, outH);
        } 
        else if (in_img.m_p_uchar1 && out_img.m_p_uchar1) {
            kernel_resize_nn_uchar1<<<gridDim, blockDim>>>(in_img.m_p_uchar1, out_img.m_p_uchar1,inW, inH, outW, outH);
        }
    }

    hipDeviceSynchronize();
}

// Example usage in main.cpp for resize:
// // Define target size or compute scale factor
// cv::Mat resized(targetW, targetH, input.type());
// CudaImg inCuda(input);
// CudaImg resCuda(resized);
// // Nearest neighbour
// cu_resize(inCuda, resCuda, false);
// cv::imshow("Resized NN", resized);
// // Bilinear interpolation
// cu_resize(inCuda, resCuda, true);
// cv::imshow("Resized Bilinear", resized);

__global__ void kernel_remove_color_channel(CudaImg t_cuda_img, int number){
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;

    if (l_x >= t_cuda_img.m_size.x || l_y >= t_cuda_img.m_size.y){ 
        return;
    }

    uchar3 here = t_cuda_img.m_p_uchar3[l_y * t_cuda_img.m_size.x + l_x];

    if (number == 1){
        here.x = 0;
    }else if (number == 2){
        here.x = 0;
        here.y = 0;
    }else if (number == 3){
        here.x = 0;
        here.y = 0;
        here.z = 0;
    }

    t_cuda_img.m_p_uchar3[l_y * t_cuda_img.m_size.x + l_x] = here;
}

void cu_run_remove_color_channel(CudaImg t_cuda_img, int number){
    hipError_t l_cerr;

    int t_block_size = 16;

    dim3 l_grid((t_cuda_img.m_size.x + t_block_size - 1) / t_block_size,

        (t_cuda_img.m_size.y + t_block_size - 1) / t_block_size);

    dim3 l_threads(t_block_size, t_block_size);

    kernel_remove_color_channel <<< l_grid, l_threads >>> (t_cuda_img, number);

    if ((l_cerr = hipGetLastError()) != hipSuccess) {

        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    }

    hipDeviceSynchronize();

}

__global__ void kernel_mirror_image(CudaImg t_src_img, CudaImg t_dst_img, int mirror_direction)
{
    // Calculate global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if within image bounds
    if (x < t_src_img.m_size.x && y < t_src_img.m_size.y)
    {
        // Calculate mirrored coordinates
        int mirrored_x = x;
        int mirrored_y = y;
        
        if (mirror_direction == 1) {
            // Horizontal mirroring (flip along y-axis)
            mirrored_x = t_src_img.m_size.x - 1 - x;
        } else if (mirror_direction == 2) {
            // Vertical mirroring (flip along x-axis)
            mirrored_y = t_src_img.m_size.y - 1 - y;
        }
        
        // Copy the pixel from source to destination (with mirroring)
        // Handle different image types (color or grayscale)
        if (t_src_img.m_p_uchar3 && t_dst_img.m_p_uchar3) {
            // RGB to RGB
            t_dst_img.at3(y, x) = t_src_img.at3(mirrored_y, mirrored_x);
        }
        else if (t_src_img.m_p_uchar1 && t_dst_img.m_p_uchar1) {
            // Grayscale to Grayscale
            t_dst_img.at1(y, x) = t_src_img.at1(mirrored_y, mirrored_x);
        }
        else if (t_dst_img.m_p_uchar3 && t_src_img.m_p_uchar1) {
            // Grayscale to RGB
            uchar1 pixel = t_src_img.at1(mirrored_y, mirrored_x);
            uchar3 color;
            color.x = color.y = color.z = pixel.x;
            t_dst_img.at3(y, x) = color;
        }
        else if (t_dst_img.m_p_uchar1 && t_src_img.m_p_uchar3) {
            // RGB to Grayscale - use a simple average for conversion
            uchar3 pixel = t_src_img.at3(mirrored_y, mirrored_x);
            uchar1 gray;
            gray.x = (pixel.x + pixel.y + pixel.z) / 3;
            t_dst_img.at1(y, x) = gray;
        }
    }
}

// Function to be called from main program - mirrors the image
void cu_mirror_image(CudaImg &t_src_img, CudaImg &t_dst_img, int mirror_direction)
{
    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (t_src_img.m_size.x + blockSize.x - 1) / blockSize.x,
        (t_src_img.m_size.y + blockSize.y - 1) / blockSize.y
    );
    
    // Launch the kernel
    kernel_mirror_image<<<gridSize, blockSize>>>(t_src_img, t_dst_img, mirror_direction);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
}