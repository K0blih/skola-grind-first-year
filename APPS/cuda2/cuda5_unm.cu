#include "hip/hip_runtime.h"
// ***********************************************************************
//
// Demo program for education in subject
// Computer Architectures and Parallel Systems.
// Petr Olivka, dep. of Computer Science, FEI, VSB-TU Ostrava, 2020/11
// email:petr.olivka@vsb.cz
//
// Example of CUDA Technology Usage wit unified memory.
// Image transformation from RGB to BW schema. 
//
// ***********************************************************************

#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>

#include "cuda_img.h"

#include "font24x40_lsb.h"

// Demo kernel to create chess board
__global__ void kernel_chessboard( CudaImg t_color_cuda_img )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    unsigned char b_or_w = 255 * ( ( blockIdx.x + blockIdx.y ) & 1 );

    // Store point into image
    t_color_cuda_img.m_p_uchar3[ l_y * t_color_cuda_img.m_size.x + l_x ] = { b_or_w, b_or_w, b_or_w };
}

void cu_create_chessboard( CudaImg t_color_cuda_img, int t_square_size )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + t_square_size - 1 ) / t_square_size,
                   ( t_color_cuda_img.m_size.y + t_square_size - 1 ) / t_square_size );
    dim3 l_threads( t_square_size, t_square_size );
    kernel_chessboard<<< l_blocks, l_threads >>>( t_color_cuda_img );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// Demo kernel to create picture with alpha channel gradient
__global__ void kernel_alphaimg( CudaImg t_color_cuda_img, uchar3 t_color )
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if ( l_y >= t_color_cuda_img.m_size.y ) return;
    if ( l_x >= t_color_cuda_img.m_size.x ) return;

    int l_diagonal = sqrtf( t_color_cuda_img.m_size.x * t_color_cuda_img.m_size.x + t_color_cuda_img.m_size.y * t_color_cuda_img.m_size.y );
    int l_dx = l_x - t_color_cuda_img.m_size.x / 2;
    int l_dy = l_y - t_color_cuda_img.m_size.y / 2;
    int l_dxy = sqrtf( l_dx * l_dx + l_dy * l_dy ) - l_diagonal / 2;

    // Store point into image
    t_color_cuda_img.m_p_uchar4[ l_y * t_color_cuda_img.m_size.x + l_x ] =
        { t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 255 - 255 * l_dxy / ( l_diagonal / 2 ) ) };
}

void cu_create_alphaimg( CudaImg t_color_cuda_img, uchar3 t_color )
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than images
    int l_block_size = 32;
    dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
                   ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
    dim3 l_threads( l_block_size, l_block_size );
    kernel_alphaimg<<< l_blocks, l_threads >>>( t_color_cuda_img, t_color );

    if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
        printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

    hipDeviceSynchronize();
}

// Kernel for inserting small BGRA image into a 3-channel BGR target
__global__ void kernel_insertimage(CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position) {
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (l_y >= t_small_cuda_pic.m_size.y) return;
    if (l_x >= t_small_cuda_pic.m_size.x) return;
    
    int l_by = l_y + t_position.y;
    int l_bx = l_x + t_position.x;
    if (l_by >= t_big_cuda_pic.m_size.y || l_by < 0) return;
    if (l_bx >= t_big_cuda_pic.m_size.x || l_bx < 0) return;
    
    // Handle both 3-channel and 4-channel source images
    uchar3 l_bg = t_big_cuda_pic.at3(l_by, l_bx);
    uchar3 l_fg;
    float alpha = 1.0f; // Default to opaque
    
    if (t_small_cuda_pic.m_p_uchar4) {
        uchar4 fg_pixel = t_small_cuda_pic.at4(l_y, l_x);
        l_fg = make_uchar3(fg_pixel.x, fg_pixel.y, fg_pixel.z);
        alpha = fg_pixel.w / 255.0f; // Normalize alpha
    } else {
        l_fg = t_small_cuda_pic.at3(l_y, l_x);
    }
    
    // Alpha blending
    uchar3 l_result;
    l_result.x = l_fg.x * alpha + l_bg.x * (1 - alpha);
    l_result.y = l_fg.y * alpha + l_bg.y * (1 - alpha);
    l_result.z = l_fg.z * alpha + l_bg.z * (1 - alpha);
    
    t_big_cuda_pic.at3(l_by, l_bx) = l_result;
}

// Function to handle image insertion with channel detection
void cu_insertimage(CudaImg &t_big_cuda_pic, CudaImg &t_small_cuda_pic, int2 t_position)
{
    hipError_t l_cerr;
    
    // Grid creation, size of grid must be equal or greater than small image
    int l_block_size = 32;
    dim3 l_blocks((t_small_cuda_pic.m_size.x + l_block_size - 1) / l_block_size,
                 (t_small_cuda_pic.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    
    kernel_insertimage<<<l_blocks, l_threads>>>(t_big_cuda_pic, t_small_cuda_pic, t_position);
    
    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));
    
    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// t_direction = 1 -> 90° clockwise
// t_direction = -1 -> 90° counter-clockwise
__global__ void kernel_rotate90_4ch(CudaImg t_img_in, CudaImg t_img_out, int t_direction)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_img_in.m_size.y) return;
    if (l_x >= t_img_in.m_size.x) return;

    // Calculate new coordinates after rotation
    int l_new_x, l_new_y;
    
    if (t_direction == 1) {
        // 90° clockwise: (x,y) -> (y, width-1-x)
        l_new_x = l_y;
        l_new_y = t_img_in.m_size.x - 1 - l_x;
    } else {
        // 90° counter-clockwise: (x,y) -> (height-1-y, x)
        l_new_x = t_img_in.m_size.y - 1 - l_y;
        l_new_y = l_x;
    }

    // Get source pixel using at4
    uchar4 l_color = t_img_in.at4(l_y, l_x);
    
    // Place it in rotated position in output image using at4
    t_img_out.at4(l_new_y, l_new_x) = l_color;
}

// Kernel for 3-channel image rotation
__global__ void kernel_rotate90(CudaImg t_img_in, CudaImg t_img_out, int t_direction)
{
    // X,Y coordinates and check image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_img_in.m_size.y) return;
    if (l_x >= t_img_in.m_size.x) return;

    // Calculate new coordinates after rotation
    int l_new_x, l_new_y;
    
    if (t_direction == 1) {
        // 90° clockwise: (x,y) -> (y, width-1-x)
        l_new_x = l_y;
        l_new_y = t_img_in.m_size.x - 1 - l_x;
    } else {
        // 90° counter-clockwise: (x,y) -> (height-1-y, x)
        l_new_x = t_img_in.m_size.y - 1 - l_y;
        l_new_y = l_x;
    }

    // Get source pixel using at3
    uchar3 l_color = t_img_in.at3(l_y, l_x);
    
    // Place it in rotated position in output image using at3
    t_img_out.at3(l_new_y, l_new_x) = l_color;
}

// Function to handle the image rotation with channel detection
void cu_rotate90(CudaImg &t_cu_img, CudaImg &t_cu_img_rotated, int t_direction)
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than image
    int l_block_size = 32;
    dim3 l_blocks((t_cu_img.m_size.x + l_block_size - 1) / l_block_size,
                 (t_cu_img.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    
    // Check if we're dealing with a 3-channel or 4-channel image
    // This could be determined by examining the cv::Mat type or passing an additional parameter
    // For this implementation, we'll check if m_p_uchar4 is not null
    bool is_4ch = (t_cu_img.m_p_uchar4 != nullptr);
    
    if (is_4ch) {
        kernel_rotate90_4ch<<<l_blocks, l_threads>>>(t_cu_img, t_cu_img_rotated, t_direction);
    } else {
        kernel_rotate90<<<l_blocks, l_threads>>>(t_cu_img, t_cu_img_rotated, t_direction);
    }

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------
// 2. Kernel to scale a 4-channel image using bilinear interpolation
__global__ void kernel_scale_4ch(CudaImg t_img_in, CudaImg t_img_out)
{
    // X,Y coordinates and check output image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_img_out.m_size.y) return;
    if (l_x >= t_img_out.m_size.x) return;

    // Calculate scaling factors
    float l_scale_x = (float)t_img_in.m_size.x / t_img_out.m_size.x;
    float l_scale_y = (float)t_img_in.m_size.y / t_img_out.m_size.y;
    
    // Calculate source position in original image
    float l_orig_x = l_x * l_scale_x;
    float l_orig_y = l_y * l_scale_y;
    
    // Get the four surrounding pixels
    int l_x0 = (int)l_orig_x;
    int l_y0 = (int)l_orig_y;
    int l_x1 = min(l_x0 + 1, (int)t_img_in.m_size.x - 1);
    int l_y1 = min(l_y0 + 1, (int)t_img_in.m_size.y - 1);
    
    // Calculate interpolation weights
    float l_dx = l_orig_x - l_x0;
    float l_dy = l_orig_y - l_y0;
    
    // Get the four surrounding colors using at4
    uchar4 l_c00 = t_img_in.at4(l_y0, l_x0);
    uchar4 l_c10 = t_img_in.at4(l_y0, l_x1);
    uchar4 l_c01 = t_img_in.at4(l_y1, l_x0);
    uchar4 l_c11 = t_img_in.at4(l_y1, l_x1);
    
    // Interpolate to get the output color
    uchar4 l_result;
    
    // Interpolate for each channel (RGB + alpha)
    // R channel
    float l_s0_r = l_c00.x * (1 - l_dx) + l_c10.x * l_dx;
    float l_s1_r = l_c01.x * (1 - l_dx) + l_c11.x * l_dx;
    l_result.x = (unsigned char)(l_s0_r * (1 - l_dy) + l_s1_r * l_dy);
    
    // G channel
    float l_s0_g = l_c00.y * (1 - l_dx) + l_c10.y * l_dx;
    float l_s1_g = l_c01.y * (1 - l_dx) + l_c11.y * l_dx;
    l_result.y = (unsigned char)(l_s0_g * (1 - l_dy) + l_s1_g * l_dy);
    
    // B channel
    float l_s0_b = l_c00.z * (1 - l_dx) + l_c10.z * l_dx;
    float l_s1_b = l_c01.z * (1 - l_dx) + l_c11.z * l_dx;
    l_result.z = (unsigned char)(l_s0_b * (1 - l_dy) + l_s1_b * l_dy);
    
    // Alpha channel
    float l_s0_a = l_c00.w * (1 - l_dx) + l_c10.w * l_dx;
    float l_s1_a = l_c01.w * (1 - l_dx) + l_c11.w * l_dx;
    l_result.w = (unsigned char)(l_s0_a * (1 - l_dy) + l_s1_a * l_dy);
    
    // Store result in output image using at4
    t_img_out.at4(l_y, l_x) = l_result;
}

// Kernel to scale a 3-channel image using bilinear interpolation
__global__ void kernel_scale(CudaImg t_img_in, CudaImg t_img_out)
{
    // X,Y coordinates and check output image dimensions
    int l_y = blockDim.y * blockIdx.y + threadIdx.y;
    int l_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (l_y >= t_img_out.m_size.y) return;
    if (l_x >= t_img_out.m_size.x) return;

    // Calculate scaling factors
    float l_scale_x = (float)t_img_in.m_size.x / t_img_out.m_size.x;
    float l_scale_y = (float)t_img_in.m_size.y / t_img_out.m_size.y;
    
    // Calculate source position in original image
    float l_orig_x = l_x * l_scale_x;
    float l_orig_y = l_y * l_scale_y;
    
    // Get the four surrounding pixels
    int l_x0 = (int)l_orig_x;
    int l_y0 = (int)l_orig_y;
    int l_x1 = min(l_x0 + 1, (int)t_img_in.m_size.x - 1);
    int l_y1 = min(l_y0 + 1, (int)t_img_in.m_size.y - 1);
    
    // Calculate interpolation weights
    float l_dx = l_orig_x - l_x0;
    float l_dy = l_orig_y - l_y0;
    
    // Get the four surrounding colors using at3
    uchar3 l_c00 = t_img_in.at3(l_y0, l_x0);
    uchar3 l_c10 = t_img_in.at3(l_y0, l_x1);
    uchar3 l_c01 = t_img_in.at3(l_y1, l_x0);
    uchar3 l_c11 = t_img_in.at3(l_y1, l_x1);
    
    // Interpolate to get the output color
    uchar3 l_result;
    
    // Interpolate for each channel (RGB)
    // R channel
    float l_s0_r = l_c00.x * (1 - l_dx) + l_c10.x * l_dx;
    float l_s1_r = l_c01.x * (1 - l_dx) + l_c11.x * l_dx;
    l_result.x = (unsigned char)(l_s0_r * (1 - l_dy) + l_s1_r * l_dy);
    
    // G channel
    float l_s0_g = l_c00.y * (1 - l_dx) + l_c10.y * l_dx;
    float l_s1_g = l_c01.y * (1 - l_dx) + l_c11.y * l_dx;
    l_result.y = (unsigned char)(l_s0_g * (1 - l_dy) + l_s1_g * l_dy);
    
    // B channel
    float l_s0_b = l_c00.z * (1 - l_dx) + l_c10.z * l_dx;
    float l_s1_b = l_c01.z * (1 - l_dx) + l_c11.z * l_dx;
    l_result.z = (unsigned char)(l_s0_b * (1 - l_dy) + l_s1_b * l_dy);
    
    // Store result in output image using at3
    t_img_out.at3(l_y, l_x) = l_result;
}

// Function to handle the image scaling with channel detection
void cu_scale(CudaImg &t_cu_orig, CudaImg &t_cu_scaled)
{
    hipError_t l_cerr;

    // Grid creation, size of grid must be equal or greater than output image
    int l_block_size = 32;
    dim3 l_blocks((t_cu_scaled.m_size.x + l_block_size - 1) / l_block_size,
                 (t_cu_scaled.m_size.y + l_block_size - 1) / l_block_size);
    dim3 l_threads(l_block_size, l_block_size);
    
    // Check if we're dealing with a 3-channel or 4-channel image
    bool is_4ch = (t_cu_orig.m_p_uchar4 != nullptr);
    
    if (is_4ch) {
        kernel_scale_4ch<<<l_blocks, l_threads>>>(t_cu_orig, t_cu_scaled);
    } else {
        kernel_scale<<<l_blocks, l_threads>>>(t_cu_orig, t_cu_scaled);
    }

    if ((l_cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(l_cerr));

    hipDeviceSynchronize();
}

// -----------------------------------------------------------------------------------------------

// 3. Implementation of LCD font drawing
__global__ void kernel_insertChar(CudaImg big_img, int2 pos, char c, const uint32_t font[256][40]) {
    int x = threadIdx.x; // 0-23
    int y = threadIdx.y; // 0-39

    if (x >= 24 || y >= 40) return;

    uint32_t row = font[(uint8_t)c][y];

    if (row & (1 << x)) {
        int img_x = pos.x + x;
        int img_y = pos.y + y;

        if (img_x < 0 || img_y < 0 || img_x >= big_img.m_size.x || img_y >= big_img.m_size.y)
            return;
        
        if (big_img.m_p_uchar4) {
            // For 4-channel images, set black with full opacity
            big_img.at4(img_y, img_x) = make_uchar4(0, 0, 0, 255);
        }
        else if (big_img.m_p_uchar3) {
            // For 3-channel images, simple black
            big_img.at3(img_y, img_x) = make_uchar3(0, 0, 0);
        }
    }
}

void insertText(CudaImg &big_img, int2 pos, const char *text, const uint32_t font[256][40]) {
    // Allocate font data on device once
    uint32_t (*d_font)[40];
    hipMalloc(&d_font, 256 * 40 * sizeof(uint32_t));
    hipMemcpy(d_font, font, 256 * 40 * sizeof(uint32_t), hipMemcpyHostToDevice);

    CudaImg d_big_img;
    size_t img_size = big_img.m_size.x * big_img.m_size.y * (big_img.m_p_uchar3 ? sizeof(uchar3) : sizeof(uchar4));
    hipMalloc(&d_big_img.m_p_void, img_size);
    d_big_img.m_size = big_img.m_size;
    
    int2 current_pos = pos;
    while (*text) {
        // Copy image to device for each character (since it may have changed)
        hipMemcpy(d_big_img.m_p_void, big_img.m_p_void, img_size, hipMemcpyHostToDevice);

        dim3 block(24, 40);
        kernel_insertChar<<<1, block>>>(d_big_img, current_pos, *text, d_font);
        hipDeviceSynchronize();

        // Copy back after each character
        hipMemcpy(big_img.m_p_void, d_big_img.m_p_void, img_size, hipMemcpyDeviceToHost);
        
        current_pos.x += 26;
        ++text;
    }

    // Free device memory
    hipFree(d_big_img.m_p_void);
    hipFree(d_font);
}